#include "hip/hip_runtime.h"
#include <math_functions.h>  // CUDA's, not bsy's, for fabs, signbit
#include <cmath>
#include "bsy/common.hpp"
#include "bsy/util/math.hpp"

namespace bsy {

template <>
voidbsy_gpu_gemm<float>(const CBLAS_TRANSPOSE TransA,
    const CBLAS_TRANSPOSE TransB, const int M, const int N, const int K,
    const float alpha, const float* A, const float* B, const float beta,
    float* C) {
  // Note that cublas follows fortran order.
  int lda = (TransA == CblasNoTrans) ? K : M;
  int ldb = (TransB == CblasNoTrans) ? N : K;
  hipblasOperation_t cuTransA =
      (TransA == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  hipblasOperation_t cuTransB =
      (TransB == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  CUBLAS_CHECK(hipblasSgemm(Bsy::cublas_handle(), cuTransB, cuTransA,
      N, M, K, &alpha, B, ldb, A, lda, &beta, C, N));
}

template <>
voidbsy_gpu_gemm<double>(const CBLAS_TRANSPOSE TransA,
    const CBLAS_TRANSPOSE TransB, const int M, const int N, const int K,
    const double alpha, const double* A, const double* B, const double beta,
    double* C) {
  // Note that cublas follows fortran order.
  int lda = (TransA == CblasNoTrans) ? K : M;
  int ldb = (TransB == CblasNoTrans) ? N : K;
  hipblasOperation_t cuTransA =
      (TransA == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  hipblasOperation_t cuTransB =
      (TransB == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  CUBLAS_CHECK(hipblasDgemm(Bsy::cublas_handle(), cuTransB, cuTransA,
      N, M, K, &alpha, B, ldb, A, lda, &beta, C, N));
}

template <>
voidbsy_gpu_gemv<float>(const CBLAS_TRANSPOSE TransA, const int M,
    const int N, const float alpha, const float* A, const float* x,
    const float beta, float* y) {
  hipblasOperation_t cuTransA =
      (TransA == CblasNoTrans) ? HIPBLAS_OP_T : HIPBLAS_OP_N;
  CUBLAS_CHECK(hipblasSgemv(Bsy::cublas_handle(), cuTransA, N, M, &alpha,
      A, N, x, 1, &beta, y, 1));
}

template <>
voidbsy_gpu_gemv<double>(const CBLAS_TRANSPOSE TransA, const int M,
    const int N, const double alpha, const double* A, const double* x,
    const double beta, double* y) {
  hipblasOperation_t cuTransA =
      (TransA == CblasNoTrans) ? HIPBLAS_OP_T : HIPBLAS_OP_N;
  CUBLAS_CHECK(hipblasDgemv(Bsy::cublas_handle(), cuTransA, N, M, &alpha,
      A, N, x, 1, &beta, y, 1));
}

template <>
voidbsy_gpu_axpy<float>(const int N, const float alpha, const float* X,
    float* Y) {
  CUBLAS_CHECK(hipblasSaxpy(Bsy::cublas_handle(), N, &alpha, X, 1, Y, 1));
}

template <>
voidbsy_gpu_axpy<double>(const int N, const double alpha, const double* X,
    double* Y) {
  CUBLAS_CHECK(hipblasDaxpy(Bsy::cublas_handle(), N, &alpha, X, 1, Y, 1));
}

voidbsy_gpu_memcpy(const size_t N, const void* X, void* Y) {
  if (X != Y) {
    CUDA_CHECK(hipMemcpy(Y, X, N, hipMemcpyDefault));
  }
}

template <>
voidbsy_gpu_scal<float>(const int N, const float alpha, float *X) {
  CUBLAS_CHECK(hipblasSscal(Bsy::cublas_handle(), N, &alpha, X, 1));
}

template <>
voidbsy_gpu_scal<double>(const int N, const double alpha, double *X) {
  CUBLAS_CHECK(hipblasDscal(Bsy::cublas_handle(), N, &alpha, X, 1));
}

template <>
voidbsy_gpu_scal<float>(const int N, const float alpha, float* X,
                           hipStream_t str) {
  hipStream_t initial_stream;
  CUBLAS_CHECK(hipblasGetStream(Bsy::cublas_handle(), &initial_stream));
  CUBLAS_CHECK(hipblasSetStream(Bsy::cublas_handle(), str));
  CUBLAS_CHECK(hipblasSscal(Bsy::cublas_handle(), N, &alpha, X, 1));
  CUBLAS_CHECK(hipblasSetStream(Bsy::cublas_handle(), initial_stream));
}

template <>
voidbsy_gpu_scal<double>(const int N, const double alpha, double* X,
                            hipStream_t str) {
  hipStream_t initial_stream;
  CUBLAS_CHECK(hipblasGetStream(Bsy::cublas_handle(), &initial_stream));
  CUBLAS_CHECK(hipblasSetStream(Bsy::cublas_handle(), str));
  CUBLAS_CHECK(hipblasDscal(Bsy::cublas_handle(), N, &alpha, X, 1));
  CUBLAS_CHECK(hipblasSetStream(Bsy::cublas_handle(), initial_stream));
}

template <>
voidbsy_gpu_axpby<float>(const int N, const float alpha, const float* X,
    const float beta, float* Y) {
 bsy_gpu_scal<float>(N, beta, Y);
 bsy_gpu_axpy<float>(N, alpha, X, Y);
}

template <>
voidbsy_gpu_axpby<double>(const int N, const double alpha, const double* X,
    const double beta, double* Y) {
 bsy_gpu_scal<double>(N, beta, Y);
 bsy_gpu_axpy<double>(N, alpha, X, Y);
}

template <>
voidbsy_gpu_dot<float>(const int n, const float* x, const float* y,
    float* out) {
  CUBLAS_CHECK(hipblasSdot(Bsy::cublas_handle(), n, x, 1, y, 1, out));
}

template <>
voidbsy_gpu_dot<double>(const int n, const double* x, const double* y,
    double * out) {
  CUBLAS_CHECK(hipblasDdot(Bsy::cublas_handle(), n, x, 1, y, 1, out));
}

template <>
voidbsy_gpu_asum<float>(const int n, const float* x, float* y) {
  CUBLAS_CHECK(hipblasSasum(Bsy::cublas_handle(), n, x, 1, y));
}

template <>
voidbsy_gpu_asum<double>(const int n, const double* x, double* y) {
  CUBLAS_CHECK(hipblasDasum(Bsy::cublas_handle(), n, x, 1, y));
}

template <>
voidbsy_gpu_scale<float>(const int n, const float alpha, const float *x,
                            float* y) {
  CUBLAS_CHECK(hipblasScopy(Bsy::cublas_handle(), n, x, 1, y, 1));
  CUBLAS_CHECK(hipblasSscal(Bsy::cublas_handle(), n, &alpha, y, 1));
}

template <>
voidbsy_gpu_scale<double>(const int n, const double alpha, const double *x,
                             double* y) {
  CUBLAS_CHECK(hipblasDcopy(Bsy::cublas_handle(), n, x, 1, y, 1));
  CUBLAS_CHECK(hipblasDscal(Bsy::cublas_handle(), n, &alpha, y, 1));
}

template <typename Dtype>
__global__ void set_kernel(const int n, const Dtype alpha, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = alpha;
  }
}

template <typename Dtype>
voidbsy_gpu_set(const int N, const Dtype alpha, Dtype* Y) {
  if (alpha == 0) {
    CUDA_CHECK(hipMemset(Y, 0, sizeof(Dtype) * N));
    return;
  }
  // NOLINT_NEXT_LINE(whitespace/operators)
  set_kernel<Dtype><<<BSY_GET_BLOCKS(N),BSY_CUDA_NUM_THREADS>>>(
      N, alpha, Y);
}

template voidbsy_gpu_set<int>(const int N, const int alpha, int* Y);
template voidbsy_gpu_set<float>(const int N, const float alpha, float* Y);
template voidbsy_gpu_set<double>(const int N, const double alpha, double* Y);

template <typename Dtype>
__global__ void add_scalar_kernel(const int n, const Dtype alpha, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] += alpha;
  }
}

template <>
voidbsy_gpu_add_scalar(const int N, const float alpha, float* Y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  add_scalar_kernel<float><<<BSY_GET_BLOCKS(N),BSY_CUDA_NUM_THREADS>>>(
      N, alpha, Y);
}

template <>
voidbsy_gpu_add_scalar(const int N, const double alpha, double* Y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  add_scalar_kernel<double><<<BSY_GET_BLOCKS(N),BSY_CUDA_NUM_THREADS>>>(
      N, alpha, Y);
}

template <typename Dtype>
__global__ void add_kernel(const int n, const Dtype* a,
    const Dtype* b, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = a[index] + b[index];
  }
}

template <>
voidbsy_gpu_add<float>(const int N, const float* a, const float* b,
    float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  add_kernel<float><<<BSY_GET_BLOCKS(N),BSY_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <>
voidbsy_gpu_add<double>(const int N, const double* a, const double* b,
    double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  add_kernel<double><<<BSY_GET_BLOCKS(N),BSY_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <typename Dtype>
__global__ void sub_kernel(const int n, const Dtype* a,
    const Dtype* b, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = a[index] - b[index];
  }
}

template <>
voidbsy_gpu_sub<float>(const int N, const float* a, const float* b,
    float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  sub_kernel<float><<<BSY_GET_BLOCKS(N),BSY_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <>
voidbsy_gpu_sub<double>(const int N, const double* a, const double* b,
    double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  sub_kernel<double><<<BSY_GET_BLOCKS(N),BSY_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <typename Dtype>
__global__ void mul_kernel(const int n, const Dtype* a,
    const Dtype* b, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = a[index] * b[index];
  }
}

template <>
voidbsy_gpu_mul<float>(const int N, const float* a,
    const float* b, float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  mul_kernel<float><<<BSY_GET_BLOCKS(N),BSY_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <>
voidbsy_gpu_mul<double>(const int N, const double* a,
    const double* b, double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  mul_kernel<double><<<BSY_GET_BLOCKS(N),BSY_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <typename Dtype>
__global__ void div_kernel(const int n, const Dtype* a,
    const Dtype* b, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = a[index] / b[index];
  }
}

template <>
voidbsy_gpu_div<float>(const int N, const float* a,
    const float* b, float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  div_kernel<float><<<BSY_GET_BLOCKS(N),BSY_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <>
voidbsy_gpu_div<double>(const int N, const double* a,
    const double* b, double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  div_kernel<double><<<BSY_GET_BLOCKS(N),BSY_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <typename Dtype>
__global__ void abs_kernel(const int n, const Dtype* a, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = abs(a[index]);
  }
}

template <>
voidbsy_gpu_abs<float>(const int N, const float* a, float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  abs_kernel<float><<<BSY_GET_BLOCKS(N),BSY_CUDA_NUM_THREADS>>>(
      N, a, y);
}

template <>
voidbsy_gpu_abs<double>(const int N, const double* a, double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  abs_kernel<double><<<BSY_GET_BLOCKS(N),BSY_CUDA_NUM_THREADS>>>(
      N, a, y);
}


template <typename Dtype>
__global__ void exp_kernel(const int n, const Dtype* a, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = exp(a[index]);
  }
}

template <>
voidbsy_gpu_exp<float>(const int N, const float* a, float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  exp_kernel<float><<<BSY_GET_BLOCKS(N),BSY_CUDA_NUM_THREADS>>>(
      N, a, y);
}

template <>
voidbsy_gpu_exp<double>(const int N, const double* a, double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  exp_kernel<double><<<BSY_GET_BLOCKS(N),BSY_CUDA_NUM_THREADS>>>(
      N, a, y);
}

template <typename Dtype>
__global__ void log_kernel(const int n, const Dtype* a, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = log(a[index]);
  }
}

template <>
voidbsy_gpu_log<float>(const int N, const float* a, float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  log_kernel<float><<<BSY_GET_BLOCKS(N),BSY_CUDA_NUM_THREADS>>>(
      N, a, y);
}

template <>
voidbsy_gpu_log<double>(const int N, const double* a, double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  log_kernel<double><<<BSY_GET_BLOCKS(N),BSY_CUDA_NUM_THREADS>>>(
      N, a, y);
}

template <typename Dtype>
__global__ void powx_kernel(const int n, const Dtype* a,
    const Dtype alpha, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = pow(a[index], alpha);
  }
}

template <>
voidbsy_gpu_powx<float>(const int N, const float* a,
    const float alpha, float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  powx_kernel<float><<<BSY_GET_BLOCKS(N),BSY_CUDA_NUM_THREADS>>>(
      N, a, alpha, y);
}

template <>
voidbsy_gpu_powx<double>(const int N, const double* a,
    const double alpha, double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  powx_kernel<double><<<BSY_GET_BLOCKS(N),BSY_CUDA_NUM_THREADS>>>(
      N, a, alpha, y);
}

template <typename Dtype>
__global__ void sqrt_kernel(const int n, const Dtype* a, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = sqrt(a[index]);
  }
}

template <>
voidbsy_gpu_sqrt<float>(const int N, const float* a, float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  sqrt_kernel<float><<<BSY_GET_BLOCKS(N),BSY_CUDA_NUM_THREADS>>>(
      N, a, y);
}

template <>
voidbsy_gpu_sqrt<double>(const int N, const double* a, double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  sqrt_kernel<double><<<BSY_GET_BLOCKS(N),BSY_CUDA_NUM_THREADS>>>(
      N, a, y);
}

DEFINE_AND_INSTANTIATE_GPU_UNARY_FUNC(sign, y[index] = (Dtype(0) < x[index])
                                      - (x[index] < Dtype(0)));
DEFINE_AND_INSTANTIATE_GPU_UNARY_FUNC(sgnbit, y[index] = signbit(x[index]));

voidbsy_gpu_rng_uniform(const int n, unsigned int* r) {
  CURAND_CHECK(hiprandGenerate(Bsy::curand_generator(), r, n));
}

template <>
voidbsy_gpu_rng_uniform<float>(const int n, const float a, const float b,
                                  float* r) {
  CURAND_CHECK(hiprandGenerateUniform(Bsy::curand_generator(), r, n));
  const float range = b - a;
  if (range != static_cast<float>(1)) {
   bsy_gpu_scal(n, range, r);
  }
  if (a != static_cast<float>(0)) {
   bsy_gpu_add_scalar(n, a, r);
  }
}

template <>
voidbsy_gpu_rng_uniform<double>(const int n, const double a, const double b,
                                   double* r) {
  CURAND_CHECK(hiprandGenerateUniformDouble(Bsy::curand_generator(), r, n));
  const double range = b - a;
  if (range != static_cast<double>(1)) {
   bsy_gpu_scal(n, range, r);
  }
  if (a != static_cast<double>(0)) {
   bsy_gpu_add_scalar(n, a, r);
  }
}

template <>
voidbsy_gpu_rng_gaussian(const int n, const float mu, const float sigma,
                            float* r) {
  CURAND_CHECK(
      hiprandGenerateNormal(Bsy::curand_generator(), r, n, mu, sigma));
}

template <>
voidbsy_gpu_rng_gaussian(const int n, const double mu, const double sigma,
                            double* r) {
  CURAND_CHECK(
      hiprandGenerateNormalDouble(Bsy::curand_generator(), r, n, mu, sigma));
}

}  // namespacebsy
